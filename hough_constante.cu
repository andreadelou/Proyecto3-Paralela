#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Modified by   : Isabel Solano, Christopher García, Andrea Lam
 Version       : 2.0
 Last modified : November 2023
 License       : Released under the GNU GPL 3.0
 Description   : Constan hough version
 To build use  : make
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <iostream>
#include <opencv2/opencv.hpp>
#include "common/pgm.h"

const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;

//*****************************************************************
// The CPU function returns a pointer to the accummulator
void CPU_HoughTran (unsigned char *pic, int w, int h, int **acc)
{
  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;  //(w^2 + h^2)/2, radio max equivalente a centro -> esquina
  *acc = new int[rBins * degreeBins];            //el acumulador, conteo depixeles encontrados, 90*180/degInc = 9000
  memset (*acc, 0, sizeof (int) * rBins * degreeBins); //init en ceros
  int xCent = w / 2;
  int yCent = h / 2;
  float rScale = 2 * rMax / rBins;

  for (int i = 0; i < w; i++) //por cada pixel
    for (int j = 0; j < h; j++) //...
      {
        int idx = j * w + i;
        if (pic[idx] > 0) //si pasa thresh, entonces lo marca
          {
            int xCoord = i - xCent;
            int yCoord = yCent - j;  // y-coord has to be reversed
            float theta = 0;         // actual angle
            for (int tIdx = 0; tIdx < degreeBins; tIdx++) //add 1 to all lines in that pixel
              {
                float r = xCoord * cos (theta) + yCoord * sin (theta);
                int rIdx = (r + rMax) / rScale;
                (*acc)[rIdx * degreeBins + tIdx]++; //+1 para este radio r y este theta
                theta += radInc;
              }
          }
      }
}

/** Memoria constante*/
// Almacenamos todos los posibles valores de senos y cosenos
// los inicializamos en el main para pasarlos al device
__constant__ float d_Cos[degreeBins];
__constant__ float d_Sin[degreeBins];

// GPU kernel. One thread per image pixel is spawned.
// The accummulator memory needs to be allocated by the host in global memory
__global__ void GPU_HoughTran(unsigned char *pic, int w, int h, int *acc, float rMax, float rScale) {
    // ya no es necesrio pasar d_Cos ni d_Sin porque estas referencias son globales
    
    int blockID = blockIdx.x;
    int threadID = threadIdx.x;
    int gloID = blockID * blockDim.x + threadID;

    if (gloID >= w * h) return; // in case of extra threads in block

    int xCent = w / 2;
    int yCent = h / 2;

    int xCoord = gloID % w - xCent;
    int yCoord = yCent - gloID / w;

    if (pic[gloID] > 0) {
        for (int tIdx = 0; tIdx < degreeBins; tIdx++) {
            //float r = xCoord * cos(tIdx) + yCoord * sin(tIdx); //probar con esto para ver diferencia en tiempo
            // llamada a d_Cos y d_Sin globales
            float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
            int rIdx = (r + rMax) / rScale;
            //debemos usar atomic, pero que race condition hay si somos un thread por pixel? explique
            atomicAdd(&acc[rIdx * degreeBins + tIdx], 1);
        }
    }
}

// Función para dibujar las líneas más pesadas en la imagen
void drawMostProminentLines(cv::Mat& image, int *h_hough, int w, int h, int rBins, int degreeBins, float radInc) {
    // Busca la línea con el peso más grande en h_hough
    int maxWeight = -1;
    int maxRIdx = -1;
    int maxTIdx = -1;

    for (int rIdx = 0; rIdx < rBins; rIdx++) {
        for (int tIdx = 0; tIdx < degreeBins; tIdx++) {
            int index = rIdx * degreeBins + tIdx;
            int weight = h_hough[index];
            if (weight > maxWeight) {
                maxWeight = weight;
                maxRIdx = rIdx;
                maxTIdx = tIdx;
            }
        }
    }

    // Calcula los valores de theta y r correspondientes a la línea con el mayor peso
    float maxTheta = maxTIdx * radInc;
    float rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;
    float rScale = 2 * rMax / rBins;
    float maxR = (maxRIdx * rScale) - rMax;

    // Pinta la línea en la imagen
    for (int i = 0; i < w; i++) {
        int j = static_cast<int>((maxR - i * cos(maxTheta)) / sin(maxTheta));
        if (j >= 0 && j < h) {
            // Colorea el píxel en la imagen original con un color diferente (por ejemplo, rojo)
            image.at<cv::Vec3b>(j, i) = cv::Vec3b(0, 0, 255); // Rojo brillante en formato BGR
        }
    }
}

// Función para comparar los resultados y registrar discrepancias
bool compareResults(int* gpuResult, int* cpuResult, int size) {
    bool match = true;
    for (int i = 0; i < size; i++) {
        if (gpuResult[i] != cpuResult[i]) {
            match = false;
            printf("Discrepancia en el índice %d: GPU = %d, CPU = %d\n", i, gpuResult[i], cpuResult[i]);
        }
    }
    return match;
}


//*****************************************************************
int main(int argc, char **argv) {
    if (argc != 2) {
        std::cerr << "Uso: " << argv[0] << " <nombre_de_imagen.pgm>" << std::endl;
        return -1;
    }

    // Load the image using OpenCV
    cv::Mat originalImage = cv::imread(argv[1], cv::IMREAD_GRAYSCALE);

    if (originalImage.empty()) {
        std::cerr << "Error al cargar la imagen." << std::endl;
        return -1;
    }

    int w = originalImage.cols;
    int h = originalImage.rows;

    // reemplazadas por las __constant__
    // float *d_Cos;
    // float *d_Sin;

    // CPU calculation
    int *cpuResult;
    CPU_HoughTran(originalImage.data, originalImage.cols, originalImage.rows, &cpuResult);

    hipMalloc((void **)&d_Cos, sizeof(float) * degreeBins);
    hipMalloc((void **)&d_Sin, sizeof(float) * degreeBins);

    // pre-compute values to be stored
    float *pcCos = (float *)malloc(sizeof(float) * degreeBins);
    float *pcSin = (float *)malloc(sizeof(float) * degreeBins);
    float rad = 0;
    for (int i = 0; i < degreeBins; i++) {
        pcCos[i] = cos(rad);
        pcSin[i] = sin(rad);
        rad += radInc;
    }

    float rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;
    float rScale = 2 * rMax / rBins;

    hipMemcpyToSymbol(HIP_SYMBOL(d_Cos), pcCos, sizeof(float) * degreeBins, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_Sin), pcSin, sizeof(float) * degreeBins, hipMemcpyHostToDevice);

    // setup and copy data from host to device
    unsigned char *d_in, *h_in;
    int *d_hough, *h_hough;

    h_in = originalImage.data; // h_in contiene los pixeles de la imagen
    h_hough = (int *)malloc(degreeBins * rBins * sizeof(int));

    hipMalloc((void **)&d_in, sizeof(unsigned char) * w * h);
    hipMalloc((void **)&d_hough, sizeof(int) * degreeBins * rBins);
    hipMemcpy(d_in, h_in, sizeof(unsigned char) * w * h, hipMemcpyHostToDevice);
    hipMemset(d_hough, 0, sizeof(int) * degreeBins * rBins);
    
    // Marcar el inicio del tiempo de ejecución del kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // execution configuration uses a 1-D grid of 1-D blocks, each made of 256 threads
    //1 thread por pixel
    int blockNum = ceil(w * h / 256);
    GPU_HoughTran<<<blockNum, 256>>>(d_in, w, h, d_hough, rMax, rScale, d_Cos, d_Sin);

    // get results from device
    hipMemcpy(h_hough, d_hough, sizeof(int) * degreeBins * rBins, hipMemcpyDeviceToHost);

    // compare CPU and GPU results
    bool resultsMatch = compareResults(h_hough, cpuResult, degreeBins * rBins);

    if (resultsMatch) {
        printf("Los resultados coinciden entre GPU y CPU.\n");
    } else {
        printf("Los resultados difieren entre GPU y CPU.\n");
    }

    // Crea una copia de la imagen original utilizando OpenCV
    cv::Mat imageWithLines;
    cv::cvtColor(originalImage, imageWithLines, cv::COLOR_GRAY2BGR); // Convierte a imagen en color

    // Llama a la función para dibujar las líneas más pesadas
    drawMostProminentLines(imageWithLines, h_hough, w, h, rBins, degreeBins, radInc);

    // Guarda la imagen con las líneas coloreadas utilizando OpenCV
    cv::imwrite("output.png", imageWithLines);

    // Marcar el final del tiempo de ejecución del kernel
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0.0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Tiempo de ejecución del kernel: %f ms\n", milliseconds);

    printf("Done!\n");

    free(pcCos);
    free(pcSin);
    free(h_hough);
    free(cpuResult);
    hipFree(d_Cos);
    hipFree(d_Sin);
    hipFree(d_in);
    hipFree(d_hough);

    return 0;
}

