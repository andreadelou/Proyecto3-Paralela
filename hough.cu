#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : make
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "common/pgm.h"

const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;

void CPU_HoughTran(unsigned char *pic, int w, int h, int **acc) {
    float rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;
    *acc = new int[rBins * degreeBins];
    memset(*acc, 0, sizeof(int) * rBins * degreeBins);
    int xCent = w / 2;
    int yCent = h / 2;
    float rScale = 2 * rMax / rBins;

    for (int i = 0; i < w; i++) {
        for (int j = 0; j < h; j++) {
            int idx = j * w + i;
            if (pic[idx] > 0) {
                int xCoord = i - xCent;
                int yCoord = yCent - j;
                float theta = 0;
                for (int tIdx = 0; tIdx < degreeBins; tIdx++) {
                    float r = xCoord * cos(theta) + yCoord * sin(theta);
                    int rIdx = (r + rMax) / rScale;
                    (*acc)[rIdx * degreeBins + tIdx]++;
                    theta += radInc;
                }
            }
        }
    }
}

__global__ void GPU_HoughTran(unsigned char *pic, int w, int h, int *acc, float rMax, float rScale, float *d_Cos, float *d_Sin) {
    int blockID = blockIdx.x;
    int threadID = threadIdx.x;
    int gloID = blockID * blockDim.x + threadID;

    if (gloID >= w * h) return;

    int xCent = w / 2;
    int yCent = h / 2;
    int xCoord = gloID % w - xCent;
    int yCoord = yCent - gloID / w;

    if (pic[gloID] > 0) {
        for (int tIdx = 0; tIdx < degreeBins; tIdx++) {
            float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
            int rIdx = (r + rMax) / rScale;
            atomicAdd(&acc[rIdx * degreeBins + tIdx], 1);
        }
    }
}

int main(int argc, char **argv) {
    int i;

    PGMImage inImg(argv[1]);

    int *cpuht;
    int w = inImg.x_dim;
    int h = inImg.y_dim;

    float *d_Cos;
    float *d_Sin;

    hipMalloc((void **)&d_Cos, sizeof(float) * degreeBins);
    hipMalloc((void **)&d_Sin, sizeof(float) * degreeBins);

    CPU_HoughTran(inImg.pixels, w, h, &cpuht);

    float *pcCos = (float *)malloc(sizeof(float) * degreeBins);
    float *pcSin = (float *)malloc(sizeof(float) * degreeBins);
    float rad = 0;
    for (i = 0; i < degreeBins; i++) {
        pcCos[i] = cos(rad);
        pcSin[i] = sin(rad);
        rad += radInc;
    }

    float rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;
    float rScale = 2 * rMax / rBins;

    hipMemcpy(d_Cos, pcCos, sizeof(float) * degreeBins, hipMemcpyHostToDevice);
    hipMemcpy(d_Sin, pcSin, sizeof(float) * degreeBins, hipMemcpyHostToDevice);

    unsigned char *d_in, *h_in;
    int *d_hough, *h_hough;

    h_in = inImg.pixels;
    h_hough = (int *)malloc(degreeBins * rBins * sizeof(int));

    hipMalloc((void **)&d_in, sizeof(unsigned char) * w * h);
    hipMalloc((void **)&d_hough, sizeof(int) * degreeBins * rBins);
    hipMemcpy(d_in, h_in, sizeof(unsigned char) * w * h, hipMemcpyHostToDevice);
    hipMemset(d_hough, 0, sizeof(int) * degreeBins * rBins);

    int blockNum = ceil(w * h / 256);
    // Marcar el inicio del tiempo de ejecución del kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    GPU_HoughTran<<<blockNum, 256>>>(d_in, w, h, d_hough, rMax, rScale, d_Cos, d_Sin);

    // Marcar el final del tiempo de ejecución del kernel
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0.0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Tiempo de ejecución del kernel: %f ms\n", milliseconds);

    hipMemcpy(h_hough, d_hough, sizeof(int) * degreeBins * rBins, hipMemcpyDeviceToHost);

    for (i = 0; i < degreeBins * rBins; i++) {
        if (cpuht[i] != h_hough[i])
            printf("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
    }
    printf("Done!\n");

    free(pcCos);
    free(pcSin);
    delete[] cpuht;
    free(h_hough);

    hipFree(d_Cos);
    hipFree(d_Sin);
    hipFree(d_in);
    hipFree(d_hough);

    return 0;
}

